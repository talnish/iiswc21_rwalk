#include "hip/hip_runtime.h"
#include "rwalk.cuh"
#include <stdio.h>
#include <assert.h>

int64_t * d_p_scan_list = NULL;
int64_t * d_v_list = NULL;
float * d_w_list = NULL;
int64_t *d_global_walk = NULL;

int tblocksize = 512;
int nblock;

void __global__ device_rwalk(
	int m_walk_length,
	int n_walks_per_node,
	int total_num_nodes, 
	unsigned long long rnumber, 
	int64_t * d_p_scan_list, int64_t * d_v_list, float * d_w_list, int64_t *d_global_walk){
		int64_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if(i >= total_num_nodes){
			return;
		}

		long long int w;
	    for(int w_n = 0; w_n < n_walks_per_node; ++w_n) {
			d_global_walk[( total_num_nodes * w_n * m_walk_length) + ( i * m_walk_length ) + 0] = i;
			// d_global_walk[( i * m_walk_length * n_walks_per_node ) + ( w_n * m_walk_length ) + 0] = i;
			float prev_time_stamp = 0;
			int64_t src_node = i;
			int walk_cnt;
			for(walk_cnt = 1; walk_cnt < m_walk_length; ++walk_cnt) {
			  int valid_neighbor_cnt = 0;
			  for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){
				if(d_w_list[w] > prev_time_stamp){
				  valid_neighbor_cnt++;
				  break;
				}
			  }
			  if(valid_neighbor_cnt == 0) {
				break;
			  }
			  float min_bound = d_w_list[d_p_scan_list[src_node]];
			  float max_bound = d_w_list[d_p_scan_list[src_node]];
			  for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){
				if(d_w_list[w] < min_bound)
				  min_bound = d_w_list[w];
				if(d_w_list[w] > max_bound)
				  max_bound = d_w_list[w];
			  }
			  float time_boundary_diff = (max_bound - min_bound);

			  if(time_boundary_diff < 0.0000001){
				for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){ // We randomly pick 1 neighbor, we just pick the first
					if(d_w_list[w] > prev_time_stamp){
						d_global_walk[( total_num_nodes * w_n * m_walk_length) + ( i * m_walk_length ) + walk_cnt] = d_v_list[w];
						src_node = d_v_list[w];
						prev_time_stamp = d_w_list[w];
						break;
					}
				}
				continue; 
			  }
			  
			  double exp_summ = 0;            
			  for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){
				if(d_w_list[w] > prev_time_stamp){
				  exp_summ += exp((float)(d_w_list[w]-prev_time_stamp)/time_boundary_diff);
				}
			  }

			  double curCDF = 0, nextCDF = 0;
			  double random_number = rnumber * 1.0 / ULLONG_MAX;
			  rnumber = rnumber * (unsigned long long)25214903917 + 11;   
			  bool fall_through = false;
			  for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){
				if(d_w_list[w] > prev_time_stamp){
					nextCDF += (exp((float)(d_w_list[w]-prev_time_stamp)/time_boundary_diff) * 1.0 / exp_summ);
					if(nextCDF >= random_number && curCDF <= random_number) {
					  d_global_walk[( total_num_nodes * w_n * m_walk_length) + ( i * m_walk_length ) + walk_cnt] = d_v_list[w];
					//   d_global_walk[( i * m_walk_length * n_walks_per_node ) + ( w_n * m_walk_length ) + walk_cnt] = d_v_list[w];
					  src_node = d_v_list[w];
					  prev_time_stamp = d_w_list[w];
					  fall_through = true;
					  break;
				  } else {
					  curCDF = nextCDF;
				  }
				}
			  }
			  if(!fall_through){
				for(w = d_p_scan_list[src_node]; w < d_p_scan_list[src_node+1]; w++){ // This line should not be reached anyway (reaching this line means something is wrong). But just for testing, we randomly pick 1 neighbor, we just pick the first
				  if(d_w_list[w] > prev_time_stamp){
					d_global_walk[( total_num_nodes * w_n * m_walk_length) + ( i * m_walk_length ) + walk_cnt] = d_v_list[w];
					// d_global_walk[( i * m_walk_length * n_walks_per_node ) + ( w_n * m_walk_length ) + walk_cnt] = d_v_list[w];
					src_node = d_v_list[w];
					prev_time_stamp = d_w_list[w];
					break; 
				  }
				}
			  }
			}
			if (walk_cnt != m_walk_length){	
			  d_global_walk[( total_num_nodes * w_n * m_walk_length) + ( i * m_walk_length ) + walk_cnt] = -1;
			}
			
		}
	}

#define cudaCheck(err) { \
	if (err != hipSuccess) { \
		printf("CUDA error: %s: %s, line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		assert(err == hipSuccess); \
	} \
}
void cleanUpGPU_rwalk(){
	cudaCheck(hipFree(d_p_scan_list));
	cudaCheck(hipFree(d_v_list));
	cudaCheck(hipFree(d_w_list));
	cudaCheck(hipFree(d_global_walk));
}

void initializeGPU_rwalk(int max_walk_length, int num_walks_per_node){
	// Device query
	int nDevices;
	cudaCheck(hipGetDeviceCount(&nDevices));
	int device = 0;
	cudaCheck(hipSetDevice(device));
	hipDeviceProp_t prop;
	cudaCheck(hipGetDeviceProperties(&prop, device));
	tblocksize = prop.maxThreadsPerBlock;
#if defined(DEBUG)
	printf(" Max Threads Per Block %d\n", tblocksize);
#endif
	nblock = (num_of_nodes - 1) / tblocksize + 1;	
	cudaCheck(hipMalloc((void**) & d_p_scan_list, (num_of_nodes + 1) * sizeof(int64_t)));

	cudaCheck(hipMalloc((void**)&d_v_list, num_of_edges * sizeof(int64_t)));

	cudaCheck(hipMalloc((void**) & d_w_list, num_of_edges * sizeof(float)));

	cudaCheck(hipMalloc((void**) & d_global_walk, num_of_nodes * max_walk_length * MAX_NUM_OF_WALK_PER_NODE * sizeof(int64_t)));

}

void TransferDataToGPU_rwalk(){
	cudaCheck(hipMemcpy( d_p_scan_list, p_scan_list,
				(num_of_nodes + 1) * sizeof(int64_t) , hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy( d_v_list, v_list,
				num_of_edges * sizeof(int64_t) , hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy( d_w_list, w_list,
				num_of_edges * sizeof(float) , hipMemcpyHostToDevice));
}

void GetResultData_rwalk(int max_walk_length, int num_walks_per_node, long long int offset){
	cudaCheck(hipMemcpy(global_walk + offset, d_global_walk, num_of_nodes * max_walk_length * num_walks_per_node * sizeof(int64_t), hipMemcpyDeviceToHost));
}

void TrainGPU_rwalk(int max_walk_length, int num_walks_per_node, unsigned long long random_number) {
	TransferDataToGPU_rwalk();
	int i;	
	for(i = 0; i < num_walks_per_node / MAX_NUM_OF_WALK_PER_NODE; i++){
		device_rwalk<<<nblock,tblocksize>>>(
			max_walk_length,
			MAX_NUM_OF_WALK_PER_NODE,
			num_of_nodes,
			random_number, 
			d_p_scan_list, d_v_list, d_w_list, d_global_walk);
		GetResultData_rwalk(max_walk_length, MAX_NUM_OF_WALK_PER_NODE, (long long int)num_of_nodes * max_walk_length * MAX_NUM_OF_WALK_PER_NODE * i);
	}

	device_rwalk<<<nblock,tblocksize>>>(
		max_walk_length,
		num_walks_per_node % MAX_NUM_OF_WALK_PER_NODE,
		num_of_nodes,
		random_number, 
		d_p_scan_list, d_v_list, d_w_list, d_global_walk);
	GetResultData_rwalk(max_walk_length, num_walks_per_node % MAX_NUM_OF_WALK_PER_NODE, (long long int)num_of_nodes * max_walk_length * MAX_NUM_OF_WALK_PER_NODE * i);
	
#if defined(DEBUG)
	cudaCheck(hipGetLastError());
	cudaCheck(hipDeviceSynchronize());
#endif

}

