#include "hip/hip_runtime.h"
#include "word2vec.cuh"
#include <stdio.h>
#include <assert.h>

__constant__ real expTable_c[EXP_TABLE_SIZE];

real * d_syn0 = NULL;
real * d_syn1neg = NULL;
int  * d_sen = NULL;
int  * d_sen_length = NULL;
unsigned int * d_random = NULL;
int * d_table = NULL;

int maxThreadsPerBlock = 1024;
int shared_mem_usage;

void __global__ device_memset(real * array, int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		array[idx] = 0;
}


__device__ void reduceInWarp(float * f, int idInWarp){
	#if defined PARALLEL_RED && PARALLEL_RED == 1
		#if defined THREADS_PER_WORD && THREADS_PER_WORD <= 32
			#if defined THREADS_PER_WORD && THREADS_PER_WORD == 32	
			f[idInWarp] += f[idInWarp + 16];
			#endif		
			#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 16	
			f[idInWarp] += f[idInWarp + 8];
			#endif	
			#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 8	
			f[idInWarp] += f[idInWarp + 4];
			#endif	
			#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 4	
			f[idInWarp] += f[idInWarp + 2];
			#endif	
			#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 2	
			f[idInWarp] += f[idInWarp + 1];
			#endif
		#else	
		for (unsigned int i=THREADS_PER_WORD /2; i>32; i>>=1) {
			if (idInWarp < i) {
				f[idInWarp] += f[idInWarp + i];
			}
			__syncthreads();
		}
		if (idInWarp < 32){
			f[idInWarp] += f[idInWarp + 32];
			f[idInWarp] += f[idInWarp + 16];
			f[idInWarp] += f[idInWarp + 8];
			f[idInWarp] += f[idInWarp + 4];
			f[idInWarp] += f[idInWarp + 2];
			f[idInWarp] += f[idInWarp + 1];
		}
		#endif
	#else
		if (idInWarp == 0){
			// float sum = 0;
			for(int i = 1; i < THREADS_PER_WORD; i++){
				f[idInWarp] += f[idInWarp + i];
			}
			// f[0] = sum;
		}
		__syncthreads();
	#endif
}

void __global__ device_cbow(long id, int layer1_size, int layer1_size_aligned,
		int window, int negative, int table_size, int vocab_size,
		int * d_sen, int * d_table,
		volatile float * d_syn0, volatile float *d_syn1neg,
		// float * d_syn0, float *d_syn1neg,
		unsigned int * d_random, int * d_sen_length, float alpha, int syn0_size){

    int batch_id = blockIdx.y;
	int sentence_position = (threadIdx.x / THREADS_PER_WORD) + (blockDim.x / THREADS_PER_WORD) * blockIdx.x;
	int idInWarp = threadIdx.x % THREADS_PER_WORD;

	extern __shared__ float shared[];
	float * f = shared + (threadIdx.x / THREADS_PER_WORD) * THREADS_PER_WORD;
	// float * neu1 = shared + BLOCK_SIZE + (threadIdx.x / THREADS_PER_WORD) * layer1_size_aligned;
	float * neu1e= shared + BLOCK_SIZE + (blockDim.x / THREADS_PER_WORD) * layer1_size_aligned + (threadIdx.x / THREADS_PER_WORD) * layer1_size_aligned;

	if (sentence_position < d_sen_length[batch_id]) {
		unsigned long long next_random = d_random[batch_id * MAX_SENTENCE_LENGTH + sentence_position];

		// for (int sentence_idx = 0; sentence_idx < sentence_num; sentence_idx++){

			for (int c = idInWarp; c < layer1_size; c+=THREADS_PER_WORD) neu1e[c] = 0;

			next_random = next_random * (unsigned long long)25214903917 + 11;
			int b = next_random % window;
			int word = d_sen[batch_id * MAX_SENTENCE_LENGTH + sentence_position];
			// in -> hidden
			// int cw = 0;
			for (int a = b; a < window * 2 + 1 - b; a++){
				if (a != window) {
					int w = sentence_position - window + a;
					if (w < 0)
						continue;
					if (w>= d_sen_length[batch_id])
						continue;
					int last_word = d_sen[batch_id * MAX_SENTENCE_LENGTH + w];
					int l1 = last_word * layer1_size_aligned;
					for (int c = idInWarp; c < layer1_size; c+= THREADS_PER_WORD)
						// neu1[c] += d_syn0[c + last_word * layer1_size_aligned];
						neu1e[c] = 0;

			// 		cw++;
			// 	}
			
			// if (cw) {
				// for (int c = idInWarp; c < layer1_size; c+= THREADS_PER_WORD)
				// 	neu1[c] /= cw;
			
			// NEGATIVE SAMPLING
			int target, label;
				for (int d = 0; d < negative + 1; d++) {


					if (d == 0) {
						target = word;
						label = 1;
					} else {
						next_random = next_random * (unsigned long long)25214903917 + 11;
						target = d_table[(next_random >> 16) % table_size];
						if (target == 0)
							target = next_random % (vocab_size - 1) + 1;
						if (target == word)
							continue;
						label = 0;
					}
					int l2 = target * layer1_size_aligned;
					f[idInWarp] = 0;
				
					
					for (int c = idInWarp; c < layer1_size; c+=THREADS_PER_WORD){
						// f[idInWarp] += neu1[c] * d_syn1neg[c + l2];   
						f[idInWarp] += d_syn0[c + l1] * d_syn1neg[c + l2];   
					}
					
					#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 64
					__syncthreads();
					#endif
					// Do reduction here;
					reduceInWarp(f, idInWarp);

					#if defined THREADS_PER_WORD && THREADS_PER_WORD >= 64
					__syncthreads();
					#endif
					
					float g;
					if (f[0] > MAX_EXP)
						g = (label - 1) * alpha;
					else if (f[0] < -MAX_EXP)
						g = (label - 0) * alpha;
					else
						g = (label - expTable_c[(int) ((f[0] + MAX_EXP)
									* (EXP_TABLE_SIZE / MAX_EXP / 2))]) * alpha;

					//__syncthreads();	
					for (int c = idInWarp; c < layer1_size; c+=THREADS_PER_WORD)
						neu1e[c] += g * d_syn1neg[c + l2];
					for (int c = idInWarp; c < layer1_size; c+=THREADS_PER_WORD)
						// d_syn1neg[c + l2] += g * neu1[c];
						d_syn1neg[c + l2] += g * d_syn0[c + l1];
					
				}
			// hidden -> in
			// for (int a = b; a < window * 2 + 1 - b; a++)
			// 	if (a != window) {
			// 		int w = sentence_position - window + a;
			// 		if (w < 0)
			// 			continue;
			// 		if (w >= MAX_SENTENCE_LENGTH)
			// 			continue;
			// 		int last_word = d_sen[sentence_idx * MAX_SENTENCE_LENGTH + w];

					for (int c = idInWarp; c < layer1_size; c+=THREADS_PER_WORD)
						d_syn0[c + l1] += neu1e[c];

				}
			}
			__threadfence_block();
			// }
		// }// End for sentence_idx
		// Update d_random
		if (idInWarp == 0 ) d_random[batch_id * MAX_SENTENCE_LENGTH + sentence_position] = next_random;
	}
}

#define cudaCheck(err) { \
	if (err != hipSuccess) { \
		printf("CUDA error: %s: %s, line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		assert(err == hipSuccess); \
	} \
}
void cleanUpGPU(){
	cudaCheck(hipFree(d_syn1neg));
	// cudaCheck(hipFree(d_avg_syn1neg));
	cudaCheck(hipFree(d_syn0));
	// cudaCheck(hipFree(d_avg_syn0));
	cudaCheck(hipHostFree(sen));
	cudaCheck(hipFree(d_sen));
	cudaCheck(hipHostFree(sen_length));
	cudaCheck(hipFree(d_sen_length));
	cudaCheck(hipFree(d_random));
	cudaCheck(hipFree(d_table));
}
void initializeGPU(){
	// Device query
	int nDevices;
	cudaCheck(hipGetDeviceCount(&nDevices));
	int device = 0;
	cudaCheck(hipSetDevice(device));
	hipDeviceProp_t prop;
	cudaCheck(hipGetDeviceProperties(&prop, device));
	maxThreadsPerBlock = prop.maxThreadsPerBlock;

	real * h_expTable = (real *)malloc((EXP_TABLE_SIZE ) * sizeof(real));
	for (int i = 0; i < EXP_TABLE_SIZE; i++) {
		h_expTable[i] = exp((i / (real)EXP_TABLE_SIZE * 2 - 1) * MAX_EXP);
		h_expTable[i] = h_expTable[i] / (h_expTable[i] + 1);
	}
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(expTable_c), h_expTable, sizeof(real) * EXP_TABLE_SIZE));
	free(h_expTable);

	if (negative>0) {
		int syn1neg_size = vocab_size * layer1_size_aligned;
		cudaCheck(hipMalloc((void**) & d_syn1neg, syn1neg_size * sizeof(real)));
		// call memset kernel
		device_memset<<<syn1neg_size / maxThreadsPerBlock + 1, maxThreadsPerBlock>>>(d_syn1neg, syn1neg_size);
		cudaCheck(hipGetLastError());
		cudaCheck(hipDeviceSynchronize());

	}

	int syn0_size = vocab_size * layer1_size_aligned;
	
	cudaCheck(hipMalloc((void**) & d_syn0, syn0_size * sizeof(real)));
	cudaCheck(hipMemcpy(d_syn0, syn0, syn0_size * sizeof(real), hipMemcpyHostToDevice));

	cudaCheck(hipHostMalloc((void**)&sen, (MAX_SENTENCE_NUM * BATCH_SYN * MAX_SENTENCE_LENGTH) * sizeof(int) ));
	cudaCheck(hipMalloc((void**)& d_sen, (BATCH_SYN * MAX_SENTENCE_LENGTH) * sizeof(int) ));

	cudaCheck(hipHostMalloc((void**)&sen_length, (MAX_SENTENCE_NUM * BATCH_SYN) * sizeof(int) ));
	cudaCheck(hipMalloc((void**)& d_sen_length, (BATCH_SYN) * sizeof(int) ));

	cudaCheck(hipMalloc((void**) & d_random, BATCH_SYN * MAX_SENTENCE_LENGTH * sizeof(unsigned int)));
	int h_random[BATCH_SYN * MAX_SENTENCE_LENGTH];
	for (int i = 0 ; i < BATCH_SYN * MAX_SENTENCE_LENGTH; i++) h_random[i] = (unsigned int) rand();
	cudaCheck(hipMemcpy(d_random, h_random, BATCH_SYN * MAX_SENTENCE_LENGTH * sizeof(unsigned int), hipMemcpyHostToDevice));

	cudaCheck(hipMalloc((void**) & d_table, table_size * sizeof(int)));
	hipMemcpy(d_table, table, table_size * sizeof(int), hipMemcpyHostToDevice);

	shared_mem_usage = (BLOCK_SIZE + (BLOCK_SIZE/THREADS_PER_WORD) * layer1_size_aligned * 2) * sizeof(real);

}

void TransferDataToGPU(long id){
	cudaCheck(hipMemcpy( d_sen , sen + (int) id * MAX_SENTENCE_LENGTH * BATCH_SYN,
				(BATCH_SYN * MAX_SENTENCE_LENGTH) * sizeof(int) , hipMemcpyHostToDevice));

	cudaCheck(hipMemcpy( d_sen_length , sen_length + (int) id * BATCH_SYN,
				(BATCH_SYN) * sizeof(int) , hipMemcpyHostToDevice));
}

void GetResultData(){
	cudaCheck(hipMemcpy(syn0, d_syn0, vocab_size * layer1_size_aligned * sizeof(real), hipMemcpyDeviceToHost));
}

void TrainGPU(long id, float alpha) {
	TransferDataToGPU(id);
	int syn0_size = vocab_size * layer1_size_aligned;

	dim3 numBlock(MAX_SENTENCE_LENGTH / (BLOCK_SIZE/THREADS_PER_WORD) + 1, BATCH_SYN);
	device_cbow<<<numBlock,BLOCK_SIZE, shared_mem_usage >>>(id, layer1_size, layer1_size_aligned, window,
			 negative, table_size,  vocab_size,	 d_sen, d_table, d_syn0, d_syn1neg, d_random, d_sen_length, alpha, syn0_size);
	
#if defined(DEBUG)
	cudaCheck(hipGetLastError());
	cudaCheck(hipDeviceSynchronize());
#endif

}

